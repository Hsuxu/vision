#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <torch/library.h>
#include <THC/THCAtomics.cuh>

#include "cuda_helpers.h"

namespace vision {
namespace ops {

namespace {

template <typename T>
__global__ void roi_pool3d_forward_kernel_impl(
    int nthreads,
    const T* input,
    const T spatial_scale,
    int channels,
    int depth,
    int height,
    int width,
    int pooled_depth,
    int pooled_height,
    int pooled_width,
    const T* rois,
    T* output,
    int* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int pd = (index / pooled_width / pooled_height) % pooled_depth; 
    int c = (index / pooled_width / pooled_height / pooled_depth) % channels;
    int n = index / pooled_width / pooled_height /pooled_depth / channels;

    const T* offset_rois = rois + n * 7;
    int roi_batch_ind = offset_rois[0];
    int roi_start_d = round(offset_rois[1] * spatial_scale);
    int roi_start_w = round(offset_rois[2] * spatial_scale);
    int roi_start_h = round(offset_rois[3] * spatial_scale);
    int roi_end_d = round(offset_rois[4] * spatial_scale);
    int roi_end_w = round(offset_rois[5] * spatial_scale);
    int roi_end_h = round(offset_rois[6] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_depth = max(roi_end_d - roi_start_d + 1, 1);
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    T bin_size_d = static_cast<T>(roi_depth) / static_cast<T>(pooled_depth);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<T>(pd) * bin_size_d));
    int hstart = static_cast<int>(floor(static_cast<T>(ph) * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw) * bin_size_w));
    int dend = static_cast<int>(ceil(static_cast<T>(pd + 1) * bin_size_d));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1) * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1) * bin_size_w));

    // Add roi offsets and clip to input boundaries
    dstart = min(max(dstart + roi_start_d, 0), depth);
    dend = min(max(dend + roi_start_d, 0), depth);
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart)|| (dend <= dstart);

    // Define an empty pooling region to be zero
    T maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    const T* offset_input = input + (roi_batch_ind * channels + c) * height * width * depth;
    for (int d = dstart; h < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          int input_index = d * width *height + h * width + w;
          if (offset_input[input_index] > maxval) {
            maxval = offset_input[input_index];
            maxidx = input_index;
          }
        }
      }
    }
    output[index] = maxval;
    argmax_data[index] = maxidx;
  }
}

template <typename T>
__global__ void roi_pool3d_backward_kernel_impl(
    int nthreads,
    const T* grad_output,
    const int* argmax_data,
    int num_rois,
    const T spatial_scale,
    int channels,
    int depth,
    int height,
    int width,
    int pooled_depth,
    int pooled_height,
    int pooled_width,
    T* grad_input,
    const T* rois,
    int n_stride,
    int c_stride,
    int d_stride,
    int h_stride,
    int w_stride) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int pd = (index / pooled_width / pooled_height) % pooled_depth;
    int c = (index / pooled_width / pooled_height /pooled_depth) % channels;
    int n = index / pooled_width / pooled_height /pooled_depth / channels;

    const T* offset_rois = rois + n * 7;
    int roi_batch_ind = offset_rois[0];
    T* grad_input_offset = grad_input + ((roi_batch_ind * channels + c) * height * width * depth);

    int output_offset = n * n_stride + c * c_stride;
    const int* argmax_data_offset = argmax_data + (n * channels + c) * pooled_height * pooled_width * pooled_depth;
    int argmax = argmax_data_offset[pd * pooled_height*pooled_width + ph * pooled_width + pw];

    if (argmax != -1) {
      atomicAdd(
          grad_input_offset + argmax,
          static_cast<T>(
              grad_output[output_offset + ph * h_stride + pw * w_stride + pd * d_stride]));
    }
  }
}

std::tuple<at::Tensor, at::Tensor> roi_pool3d_forward_kernel(
    const at::Tensor& input,
    const at::Tensor& rois,
    double spatial_scale,
    int64_t pooled_depth,
    int64_t pooled_height,
    int64_t pooled_width) {
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
  TORCH_CHECK(rois.is_cuda(), "rois must be a CUDA tensor");
  TORCH_CHECK(rois.size(1) == 7, "Tensor rois should have shape as Tensor[K, 7]");

  at::TensorArg input_t{input, "input", 1}, rois_t{rois, "rois", 2};

  at::CheckedFrom c = "roi_pool3d_forward_kernel";
  at::checkAllSameGPU(c, {input_t, rois_t});
  at::checkAllSameType(c, {input_t, rois_t});

  at::cuda::CUDAGuard device_guard(input.device());

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto depth = input.size(2);
  auto height = input.size(3);
  auto width = input.size(4);

  at::Tensor output = at::zeros({num_rois, channels, pooled_depth, pooled_height, pooled_width}, input.options());
  at::Tensor argmax = at::zeros({num_rois, channels, pooled_depth, pooled_height, pooled_width}, input.options().dtype(at::kInt));

  auto output_size = num_rois * pooled_depth * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      ceil_div(static_cast<int64_t>(output_size), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  if (output.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(output, argmax);
  }

  auto input_ = input.contiguous(), rois_ = rois.contiguous();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "roi_pool3d_forward_kernel", [&] {
        roi_pool_forward_kernel_impl<scalar_t><<<grid, block, 0, stream>>>(
            output_size,
            input_.data_ptr<scalar_t>(),
            spatial_scale,
            channels,
            depth,
            height,
            width,
            pooled_depth,
            pooled_height,
            pooled_width,
            rois_.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            argmax.data_ptr<int>());
      });
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(output, argmax);
}

at::Tensor roi_pool3d_backward_kernel(
    const at::Tensor& grad,
    const at::Tensor& rois,
    const at::Tensor& argmax,
    double spatial_scale,
    int64_t pooled_depth,
    int64_t pooled_height,
    int64_t pooled_width,
    int64_t batch_size,
    int64_t channels,
    int64_t depth,
    int64_t height,
    int64_t width) {
  // Check if input tensors are CUDA tensors
  TORCH_CHECK(grad.is_cuda(), "grad must be a CUDA tensor");
  TORCH_CHECK(rois.is_cuda(), "rois must be a CUDA tensor");
  TORCH_CHECK(argmax.is_cuda(), "argmax must be a CUDA tensor");

  at::TensorArg grad_t{grad, "grad", 1}, rois_t{rois, "rois", 2},
      argmax_t{argmax, "argmax", 3};

  at::CheckedFrom c = "roi_pool3d_backward_kernel";
  at::checkAllSameGPU(c, {grad_t, rois_t, argmax_t});
  at::checkAllSameType(c, {grad_t, rois_t});

  at::cuda::CUDAGuard device_guard(grad.device());

  auto num_rois = rois.size(0);

  at::Tensor grad_input = at::zeros({batch_size, channels, depth, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(ceil_div(static_cast<int64_t>(grad.numel()), static_cast<int64_t>(512)), static_cast<int64_t>(4096)));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_input;
  }

  int n_stride = grad.stride(0);
  int c_stride = grad.stride(1);
  int d_stride = grad.stride(2)
  int h_stride = grad.stride(3);
  int w_stride = grad.stride(4);

  auto argmax_ = argmax.contiguous(), rois_ = rois.contiguous();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad.scalar_type(), "roi_pool3d_backward_kernel", [&] {
        roi_pool3d_backward_kernel_impl<scalar_t><<<grid, block, 0, stream>>>(
            grad.numel(),
            grad.data_ptr<scalar_t>(),
            argmax_.data_ptr<int>(),
            num_rois,
            spatial_scale,
            channels,
            depth,
            height,
            width,
            pooled_depth,
            pooled_height,
            pooled_width,
            grad_input.data_ptr<scalar_t>(),
            rois_.data_ptr<scalar_t>(),
            n_stride,
            c_stride,
            d_stride,
            h_stride,
            w_stride);
      });
  AT_CUDA_CHECK(hipGetLastError());
  return grad_input;
}

} // namespace

TORCH_LIBRARY_IMPL(torchvision, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::roi_pool3d"),
      TORCH_FN(roi_pool3d_forward_kernel));
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::_roi_pool3d_backward"),
      TORCH_FN(roi_pool3d_backward_kernel));
}

} // namespace ops
} // namespace vision
